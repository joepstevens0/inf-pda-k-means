#include "hip/hip_runtime.h"
#include "helper_functions.h"
#include "kmeans.h"
#include <iostream>
#include <string.h>
#include <thrust/reduce.h>

struct KMeansItInput {
    const size_t numPoints;
    const size_t pointSize;
    const std::vector<double> &allData;
    std::vector<double> &centroids;
    const int numClusters;
    FileCSVWriter &centroidDebugFile;
    FileCSVWriter &clustersDebugFile;
    int numThreads;
    int numBlocks;
};

struct KMeansItOutput {
    size_t numSteps;
    std::vector<int> bestClusters;
    double bestDistSquaredSum;
    std::vector<int> clusters;
};

__device__ 
void closestCentroidForPoint(const size_t pointIndex, const size_t pointSize, 
                            const double* allData, const double* centroids, const size_t numCentroids,
                            int &newCluster, double& bestDist){
    newCluster = -1;
    bestDist = 3.40282e+38; // can only get better

    const size_t p = pointIndex * pointSize;

    // Loop over all centroid points
    for (size_t i = 0; i < numCentroids; i++) {
        double dist = 0;

        // Calculate quadratic euclidean distance between data point(1) and
        // centroid point(2) [ sqrt( (x1 - x2)^2 + (y1 - y2)^2 + ...) ]
        for (size_t dim = 0; dim < pointSize; dim++)
            dist += pow(allData[p + dim] - centroids[i*pointSize + dim], 2.);

        // Change cluster index for point when distance is smaller then current
        // best
        if (dist < bestDist) {
            newCluster = i;
            bestDist = dist;
        }
    }
}

__global__
void findClosestCentroidIndexAndDistance(const size_t numPoints, const size_t pointSize, const int numClusters,
                                        const double* allData, const double* centroids, int* clusters,
                                        double* distSquaredSum, bool* changed) {
    // calc points per threads and extras
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    int pointsPerThread = numPoints/(blockDim.x*gridDim.x);
    int extraPoints = numPoints%(blockDim.x*gridDim.x);

    // Calculate point range for every thread
    int start_index =  (pointsPerThread + 1) * min(t, extraPoints) // threads with an extra point
                 + pointsPerThread * max(0, (int)t - extraPoints); // threads without an extra point
    int end_index = min(numPoints,(size_t)start_index + pointsPerThread + (t < extraPoints? 1 : 0));


    // Go over point range (every thread)
    for (int pointIndex = start_index; pointIndex < end_index; pointIndex++) {
        int newCluster = -1;
        double dist;

        closestCentroidForPoint(pointIndex, pointSize, allData, centroids, numClusters, newCluster, dist);

        distSquaredSum[t] += dist;

        if (newCluster != clusters[pointIndex]) {
            clusters[pointIndex] = newCluster;
            *changed = true;
        }
    }
}

__global__
void resetChangedAndDist(bool* changed, double* distSquaredSum) {
    *changed = false;
    memset(distSquaredSum, 0, sizeof(distSquaredSum));
}

void moveCentroidsToAverage(const size_t numPoints, const size_t pointSize, const size_t numCentroids,
                            double* centroids, int* clusters, double* allData) {

    // reset all centroids to 0
    for (int i = 0; i < numCentroids*pointSize; ++i)
        centroids[i] = 0;

    // set per centroid point counters to 0
    int pointCounts[numCentroids];
    memset(pointCounts, 0, numCentroids * sizeof(int));

    // Loop over all points of dataset
    for (size_t index = 0; index < numPoints; index++) {

        const int c = clusters[index];

        // add all dimensions to the centroid
        const size_t p = index * pointSize;
        for (int dim = 0; dim < pointSize; ++dim)
            centroids[c*pointSize + dim] += allData[p + dim];
        pointCounts[c] += 1;
    }

    // average out the centroids
    for (int i = 0; i < numCentroids; ++i) {
        if (pointCounts[i] > 0)
            for (size_t dim = 0; dim < pointSize; dim++)
                centroids[i*pointSize + dim] /= pointCounts[i];
    }
}


int kmeansCUDAIteration(KMeansItOutput &out, KMeansItInput &in, 
                        double* cuAllData, double* cuCentroids, 
                        int* cuClusters, double* cuDistSquaredSum, bool* cuChanged) {

    bool changed = true;
    std::vector<double> distSquaredSum(in.numBlocks*in.numThreads);
    out.numSteps = 0;

    // Reset clusters/centroids every iteration
    hipMemcpy(cuClusters, out.clusters.data(), out.clusters.size()*sizeof(int), hipMemcpyHostToDevice);
    

    while (changed) {
        // Reset variables
        changed = false;
        std::fill(distSquaredSum.begin(), distSquaredSum.end(), 0);
        hipMemcpy(cuCentroids, in.centroids.data(), in.centroids.size()*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(cuChanged, &changed, sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(cuDistSquaredSum, distSquaredSum.data(), distSquaredSum.size()*sizeof(double), hipMemcpyHostToDevice);

        findClosestCentroidIndexAndDistance<<<in.numBlocks, in.numThreads>>>(in.numPoints, in.pointSize, in.numClusters, 
                                                                                cuAllData, cuCentroids, cuClusters,
                                                                                cuDistSquaredSum, cuChanged);

        // Copy result from GPU
        hipMemcpy(distSquaredSum.data(), cuDistSquaredSum, distSquaredSum.size()*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&changed, cuChanged, sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(out.clusters.data(), cuClusters, out.clusters.size()*sizeof(int), hipMemcpyDeviceToHost);

        if (changed) {  // re-calculate the centroids based on current clustering
            moveCentroidsToAverage((size_t)in.numPoints, (size_t)in.pointSize, (size_t)in.numClusters,
                                            (double*)in.centroids.data(), (int*)out.clusters.data(), (double*)in.allData.data());
        }

        double dist = thrust::reduce(distSquaredSum.begin(), distSquaredSum.end());


        // Keep track of best clustering
        if (dist < out.bestDistSquaredSum) {
            out.bestClusters = out.clusters;
            out.bestDistSquaredSum = dist;
        }
        ++out.numSteps;
    }

    return 0;
}

KmeansOut kmeansCUDA(KMeansIn input) {
    
    // Init output struct obj
    KmeansOut out;
    out.stepsPerRepetition.resize(input.repetitions);
    out.bestDistSquaredSum = std::numeric_limits<double>::max();
    out.bestClusters = std::vector<int>(input.numPoints, -1);
    size_t it_of_best_cluster = 0;

    // Run seeded generator serial on CPU so reps has same start-centroids every time
    // (in case repetitions are run parallel)
    std::vector<std::vector<Point>> centroids_per_repetition(input.repetitions, std::vector<Point>(input.numClusters));
    std::vector<std::vector<double>> flat_centroids_per_repetition(input.repetitions);
    for (size_t r = 0; r < input.repetitions; r++) {
        chooseCentroidsAtRandomFromDataset(input.rng, input.numPoints,
                                            input.pointSize, input.allData,
                                            centroids_per_repetition[r]);
        // Flatten centroids array
        for (Point point: centroids_per_repetition[r])
            for (double i: point)
                flat_centroids_per_repetition[r].push_back(i);
    }

    // Init array-pointers for copying to GPU
    double* cuAllData;
    double* cuCentroids;
    int* cuClusters;
    double* cuDistSquaredSum;
    bool* cuChanged;

    // For memory alloc before repetitions start
    std::vector<double> distSquaredSum(input.numBlocks*input.numThreads); 

    // Init closest centroid index for every point: 'unknown'(-1)
    // (same for all repetitions)
    std::vector<int> startClusters(input.numPoints, -1);

    // Allocate memory for GPU (reuse for every rep)
    hipMalloc(&cuAllData, input.allData.size()*sizeof(double));
    hipMalloc(&cuCentroids, input.numClusters*input.pointSize*sizeof(double));
    hipMalloc(&cuClusters, startClusters.size()*sizeof(int));
    hipMalloc(&cuDistSquaredSum, distSquaredSum.size()*sizeof(double));
    hipMalloc(&cuChanged, sizeof(bool));

    // Copy usable information for all repetition to GPU
    hipMemcpy(cuAllData, input.allData.data(), input.allData.size()*sizeof(double), hipMemcpyHostToDevice);

    // Do the k-means routine a number of times, each time starting from
    // different random centroids (use Rng::pickRandomIndices), and keep
    // the best result of these repetitions.
    for (size_t r = 0; r < input.repetitions; r++) {
        
        // Create the iteration parameters
        KMeansItInput itinput{input.numPoints,       input.pointSize,
                            input.allData,           flat_centroids_per_repetition[r],
                            input.numClusters,       input.centroidDebugFile,
                            input.clustersDebugFile, input.numThreads, input.numBlocks};

        // create iteration output struct
        KMeansItOutput itoutput;
        itoutput.bestDistSquaredSum = std::numeric_limits<double>::max();
        itoutput.clusters = startClusters;
        itoutput.numSteps = 0;

        // start iteration
        kmeansCUDAIteration(itoutput, itinput, 
                            cuAllData, cuCentroids, 
                            cuClusters, cuDistSquaredSum, cuChanged);

        // update num of steps for this iteration
        out.stepsPerRepetition[r] = itoutput.numSteps;

        // (in case repetitions are run parallel)
        if (itoutput.bestDistSquaredSum <= out.bestDistSquaredSum) {

            // take the best clusters from the lowest repetition
            if (itoutput.bestDistSquaredSum != out.bestDistSquaredSum || r < it_of_best_cluster){
                out.bestClusters = itoutput.clusters;
                out.bestDistSquaredSum = itoutput.bestDistSquaredSum;
                it_of_best_cluster = r;
            }
        }
    }

    // free cuda memory
    hipFree(cuAllData);
    hipFree(cuCentroids);
    hipFree(cuClusters);
    hipFree(cuDistSquaredSum);
    hipFree(cuChanged);

    return out;
}